#include "hip/hip_runtime.h"
#include "..\base.h"

#include <stdio.h>

#define COLUMN_SIZE 1024 * 1024
#define ROW_SIZE 32
#define ARRAY_SIZE ROW_SIZE * COLUMN_SIZE
static f32 InitialArray[ARRAY_SIZE] = {0};
static f32 ResultArray[ROW_SIZE] = {0};

static constexpr void Init() {
	random_state RandomState = { 0xB40148552A2E3491ULL };
	for (u32 i = 0; i < ARRAY_SIZE; ++i) {
		InitialArray[i] = RandomFloat(&RandomState) * 2.0f - 1.0f;
	}
}

__device__ f32 ReduceAdd(f32 Value) {
	u32 Mask = 0xFFFFFFFF;
	Value += __shfl_down_sync(Mask, Value, 16);
	Value += __shfl_down_sync(Mask, Value, 8);
	Value += __shfl_down_sync(Mask, Value, 4);
	Value += __shfl_down_sync(Mask, Value, 2);
	Value += __shfl_down_sync(Mask, Value, 1);
	return Value;
}

#if 0
__global__ void AxisSum(f32 *In, uint2 ArrayDimensions, f32 *Out) {
	u32 X = threadIdx.x;
	u32 Y = threadIdx.y;

	f32 Result = 0.0f;
	for (u32 OffsetY = 0; OffsetY < ArrayDimensions.y; OffsetY += 1) {
		u32 GlobalIndex = (Y + OffsetY) * ArrayDimensions.x + X;
		Result += In[GlobalIndex];
	}
	Out[threadIdx.x] = Result;
}
#elif 0
__global__ void AxisSum(f32 *In, uint2 ArrayDimensions, f32 *Out) {

	f32 Result = 0.0f;
	for (u32 OffsetY = 0; OffsetY < ArrayDimensions.y; OffsetY += 32) {
		u32 GlobalIndex = (threadIdx.y + OffsetY) * ArrayDimensions.x + threadIdx.x;
		Result += In[GlobalIndex];
	}

	__shared__ f32 SharedValues[32][32];
	SharedValues[threadIdx.y][threadIdx.x] = Result;
	__syncthreads();

	if (threadIdx.x < 32 && threadIdx.y == 0) {
		f32 FinalResult = 0.0f;
		for (u32 OffsetY = 0; OffsetY < 32; OffsetY += 1) {
			FinalResult += SharedValues[OffsetY][threadIdx.x];
		}
		Out[threadIdx.x] = FinalResult;
	}
}
#else
__global__ void AxisSum(f32 *In, uint2 ArrayDimensions, f32 *Out) {

	f32 Result = 0.0f;
	for (u32 OffsetY = 0; OffsetY < ArrayDimensions.y; OffsetY += 32) {
		u32 GlobalY = threadIdx.y + OffsetY;
		u32 GlobalX = blockIdx.x * 32 + threadIdx.x;
		u32 GlobalIndex = GlobalY * ArrayDimensions.x + GlobalX;
		Result += In[GlobalIndex];
	}

	__shared__ f32 SharedValues[32][32];
	SharedValues[threadIdx.x][threadIdx.y] = Result;
	__syncthreads();

	f32 SharedValue = SharedValues[threadIdx.y][threadIdx.x];
	f32 FinalResult = ReduceAdd(SharedValue);
	if (threadIdx.x % 32 == 0) {
		Out[blockIdx.x * 32 + threadIdx.y] = FinalResult;
	}
}
#endif

s32 main() {
	Init();

	f32 *GPUArray1 = 0, *GPUArray2 = 0;
	hipMalloc(&GPUArray1, sizeof(InitialArray));
	hipMalloc(&GPUArray2, sizeof(ResultArray));
	hipMemcpy(GPUArray1, InitialArray, sizeof(InitialArray), hipMemcpyHostToDevice);

	dim3 ThreadDimensions(32, 32);
	AxisSum<<<ROW_SIZE / 32, ThreadDimensions>>>(GPUArray1, make_uint2(ROW_SIZE, COLUMN_SIZE), GPUArray2);
	// AxisSum<<<1, ROW_SIZE>>>(GPUArray1, make_uint2(ROW_SIZE, COLUMN_SIZE), GPUArray2);
	hipMemcpy(ResultArray, GPUArray2, sizeof(ResultArray), hipMemcpyDeviceToHost);

#if 1
	f32 ExpectedResults[ROW_SIZE] = {0};
	for (u32 Y = 0; Y < COLUMN_SIZE; Y += 1) {
		for (u32 X = 0; X < ROW_SIZE; X += 1) {
			ExpectedResults[X] += InitialArray[Y * ROW_SIZE + X];
		}
	}
	for (u32 i = 0; i < ROW_SIZE; ++i) {
		f32 Expected = ExpectedResults[i];
		f32 Actual = ResultArray[i];
		if (Expected == Actual) {
			printf(ANSI_COLOR_GREEN "Expected: %.2f | Actual: %.2f\n", Expected, Actual);
		} else {
			printf(ANSI_COLOR_RED "Expected: %.2f | Actual: %.2f\n", Expected, Actual);
		}
	}
#endif
	printf(ANSI_COLOR_RESET);
}
