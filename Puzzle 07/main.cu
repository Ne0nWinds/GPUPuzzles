#include "hip/hip_runtime.h"
#include "..\base.h"

#include <stdio.h>

#define BLOCKS 4
#define THREADS_PER_BLOCK 4
#define ARRAY_SIZE (BLOCKS * THREADS_PER_BLOCK)
static u32 InitialArray[ARRAY_SIZE] = {0};
static u32 ResultArray[ARRAY_SIZE] = {0};

void Init() {
	random_state RandomState = { 0xB40148552A2E3491ULL };
	for (u32 i = 0; i < ARRAY_SIZE; ++i) {
		InitialArray[i] = RandomInt(&RandomState) % 32;
	}
}

__global__ void Blocks2D(u32 *A, uint2 ArrayDimensions) {
	u32 X = blockIdx.x * blockDim.x + threadIdx.x;
	u32 Y = blockIdx.y * blockDim.y + threadIdx.y;

	A[Y * ArrayDimensions.x + X] += 10;
}

s32 main() {
	Init();

	u32 *GPUArray = 0;
	hipMalloc(&GPUArray, ARRAY_SIZE * sizeof(u32));
	hipMemcpy(GPUArray, InitialArray, ARRAY_SIZE * sizeof(u32), hipMemcpyHostToDevice);

	dim3 BlockCount = dim3(2, 2);
	dim3 ThreadsPerBlock = dim3(2, 2);
	Blocks2D<<<BlockCount, ThreadsPerBlock>>>(GPUArray, make_uint2(4, 4));
	hipMemcpy(ResultArray, GPUArray, ARRAY_SIZE * sizeof(u32), hipMemcpyDeviceToHost);

	for (u32 i = 0; i < ARRAY_SIZE; ++i) {
		u32 A = InitialArray[i];
		u32 ActualResult = ResultArray[i];
		u32 ExpectedResult = 10 + A;
		if (ExpectedResult == ActualResult) {
			printf(ANSI_COLOR_GREEN "%.2u + 10 = %u\n", A, ActualResult);
		} else {
			printf(ANSI_COLOR_RED "%.2u + 10 = %u\n", A, ActualResult);
		}
	}
	printf(ANSI_COLOR_RESET);
}
