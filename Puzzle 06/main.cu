#include "hip/hip_runtime.h"
#include "..\base.h"

#include <stdio.h>

static random_state RandomState = { 0xB40148552A2E3491 };
#define ARRAY_SIZE 9
static u32 InitialArray[ARRAY_SIZE] = {0};
static u32 ResultArray[ARRAY_SIZE * 2] = {0};

void InitRandomIntegers() {
	for (u32 i = 0; i < ARRAY_SIZE; ++i) {
		InitialArray[i] = i;
	}
}

__global__ void Blocks(u32 *In, u32 *Out, u32 Length) {
	u32 Index = blockIdx.x * blockDim.x + threadIdx.x;
	if (Index < Length) {
		Out[Index] = In[Index] + 10;
	}
}

s32 main() {
	InitRandomIntegers();
	puts("===");

	u32 *GPUArray1 = 0, *GPUArray2 = 0;
	hipMalloc(&GPUArray1, ARRAY_SIZE * sizeof(int));
	hipMalloc(&GPUArray2, ARRAY_SIZE * sizeof(int));
	hipMemcpy(GPUArray1, InitialArray, ARRAY_SIZE * sizeof(int), hipMemcpyHostToDevice);

	dim3 threadDimension(3, 1);
	dim3 blockDimension(4, 1);
	Blocks<<<blockDimension, threadDimension>>>(GPUArray1, GPUArray2, ARRAY_SIZE);
	hipMemcpy(ResultArray, GPUArray2, ARRAY_SIZE * sizeof(int), hipMemcpyDeviceToHost);

	puts("===");

	u32 i = 0;
	for (; i < ARRAY_SIZE; ++i) {
		u32 A = InitialArray[i];
		u32 ActualResult = ResultArray[i];
		u32 ExpectedResult = 10 + i;
		if (ExpectedResult == ActualResult) {
			printf(ANSI_COLOR_GREEN "%.2u + 10 = %u\n" ANSI_COLOR_RESET, A, ActualResult);
		} else {
			printf(ANSI_COLOR_RED "%.2u + 10 = %u\n" ANSI_COLOR_RESET, A, ActualResult);
		}
	}
}
